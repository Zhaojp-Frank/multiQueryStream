#include "hip/hip_runtime.h"
// Memory object sizes:
// 1. hash table build: 2 * 8 * RLEN + 2 * 32 * 1024 * RBUCKETS
// 2. after hash_build before hash_join: 8 * RLEN
// 3. each hash_join: 8 * S_CHUNK_LEN + 8 * RLEN + 8 * n_results
#include <stdio.h>
#include <stdlib.h>

#include "../md5/defs.h"
#include "../md5/kernels.cu"

#include "hj.cu"
// #include "hj_kernels.cu"

#ifndef CUDA_SAFE_CALL
    #define CUDA_SAFE_CALL(call) \
        do { \
            hipError_t err = call; \
            if(hipSuccess != err) { \
                fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \
                        __FILE__, __LINE__, hipGetErrorString(err)); \
                exit(EXIT_FAILURE); \
            } \
        } while (0)
    #endif

//#define NR_BUCKETS_DEFAULT	256


// number of records in R
//#define RLEN		(40 * 1024L * 1024L)
#define RLEN		(10L * 1024L * 1024L)
// max of R's keys
#define RKEY_MAX	(1024 * 256)
// seed of R's keys
#define RKEY_SEED	1
// number of buckets for R's hash table; should not be larger than RKEY_MAX
#define RBUCKETS	(1024 * 8) // must be power of 2

// max of S's keys
#define SKEY_MAX	(1024 * 256)
// seed of S's keys
#define SKEY_SEED	2

// number of records in each chunk read from S
#define S_CHUNK_LEN		(64L * 1024L)
// how many chunks to be read from S
#define S_CHUNK_CNT		5


        void save_data_to_disk(unsigned char *data, size_t size)
    {
        FILE *fout = fopen("md5.output", "w");
        if (!fout) {
            perror("Failed to create output file");
            exit(1);
        }
        fwrite(data, sizeof(unsigned char), size, fout);
        fclose(fout);
    }

    // Allocates a matrix with random float entries.
    void randomInit(float* data, int size)
    {
        for (int i = 0; i < size; ++i)
            data[i] = rand() / (float)RAND_MAX;
    }


    //float multiplication kernel called by MatMul()
    __global__ void MatMulKernel(float *A, float *C, int Width)
    {
            // Each thread computes one element of C by accumulating results into Cvalue
            float Cvalue = 0;
            int row = blockIdx.y * blockDim.y + threadIdx.y;
            int col = blockIdx.x * blockDim.x + threadIdx.x;

            for (int e = 0; e < Width; ++e)
                    Cvalue += A[row * Width + e] * A[Width * Width + e * Width + col - 1];
            C[row * Width + col] = Cvalue;
    }

    #define N 3

    enum DataType { dt_chr, dt_int, dt_flt };

    struct ElementAttr{
        enum DataType type;
        int dataSize;
        int resultSize;
    };

    struct ElementSet {
      union {
        unsigned char *chr_data;
        int           *int_data;
        float         *flt_data;   
      };
    };

    int mallocMemory(struct ElementSet *Data, enum DataType Type, int DataSize){
        switch(Type){
            case dt_chr:
                CUDA_SAFE_CALL( hipHostMalloc((void**)&Data->chr_data, sizeof(char) * DataSize) );      // host pinned
                // Data->chr_data = (unsigned char *)malloc(sizeof(char) * DataSize);
                break;
            case dt_int:
                CUDA_SAFE_CALL( hipHostMalloc((void**)&Data->int_data, sizeof(int) * DataSize) );      // host pinned
                // Data->int_data = (int *)malloc(sizeof(int) * DataSize);
                break;
            case dt_flt:
                CUDA_SAFE_CALL( hipHostMalloc((void**)&Data->flt_data, sizeof(float) * DataSize) );      // host pinned
                // Data->flt_data = (float *)malloc(sizeof(float) * DataSize);
                break;
        }
        return 0;
    }

    int mallocMemoryOnDevice(struct ElementSet *Data, enum DataType Type, int DataSize){
        switch(Type){
            case dt_chr:
            // printf("%s\n", "1-1");
                CUDA_SAFE_CALL(hipMalloc((void **)Data->chr_data, sizeof(char) * DataSize));
                break;
            case dt_int:
            // printf("%s\n", "1-2");
                CUDA_SAFE_CALL(hipMalloc((void **)Data->int_data, sizeof(int) * DataSize));
                break;
            case dt_flt:
            // printf("%s\n", "1-3");
                CUDA_SAFE_CALL(hipMalloc((void **)Data->flt_data, sizeof(float) * DataSize));
                break;
        }
        return 0;
    }

    int printElement(struct ElementSet Data, struct ElementAttr Job){
        switch(Job.type){
            case dt_chr:
                for (int j = 0; j < Job.dataSize; ++j)
                    printf("%c\t", Data.chr_data[j]);
                printf("\n");
                    break;
            case dt_int:
                for (int j = 0; j < Job.dataSize; ++j)
                    printf("%d\t", Data.int_data[j]);
                printf("\n");
                break;
            case dt_flt:
                for (int j = 0; j < Job.dataSize; ++j)
                    printf("%f\t", Data.flt_data[j]);
                printf("\n");
                break;
        }
        return 0;
    }


int read_r(record_t *r_tmp, int *rlen)
{
	// hipError_t res;

	// record_t *r_tmp = (record_t *)malloc(size_r);
	// if(!r_tmp) {
	// 	fprintf(stderr, "malloc failed for R\n");
	// 	return -1;
	// }
	// record_t *r_tmp;

	unsigned int seed = RKEY_SEED;
	for(int i = 0; i < RLEN; i++) {
		r_tmp[i].y = rand_r(&seed) % RKEY_MAX;
		r_tmp[i].x = i;
	}

	// *r = r_tmp;
	*rlen = RLEN;
	return 0;
}

// return the number of records actually read
int read_s(record_t *s, int slen, int skey_start)
{

	static unsigned int seed = SKEY_SEED;
	for(int i = 0; i < slen; i++) {
		s[i].y = rand_r(&seed) % (SKEY_MAX - skey_start) + skey_start;
		s[i].x = skey_start + i;
	}
	return slen;
}

// Assume R is the small table, upon which a hash table is built and kept in
// GPU memory. Assume S is the large table, for which data are fetched chunk
// by chunk, with one chunk, after another, joined with R.
// A problem with hash join is that, even though the joined results may be few,
// the number of buckets and sparse memory regions touched by the join may be
// plenty.
int main()
{
	hipStream_t *stream = (hipStream_t *) malloc(15 * sizeof(hipStream_t));

	record_t *h_r, *h_s[S_CHUNK_CNT];
	hash_table_t ht_r;
	int rlen, slen;
	struct timeval t1, t2, t_start, t_end;

	gettimeofday(&t_start, NULL);

	printf("Time of starting hj: %lf \n", TVAL(t_start));

	// Create cuda stream
        for (int i = 0; i < 15; ++i)
            CUDA_SAFE_CALL( hipStreamCreate(&stream[i]) );


        int n = N-1;

        int mm_block = 8;
        int width = 8 * 256;
        int height = width;

        // thread per block and block per grid for job n
        dim3 dimBlock[N],dimGrid[N];

		dimBlock[0].x = 32, dimBlock[0].y = 1, dimBlock[0].z = 1;
        dimGrid[0].x = 4096 / dimBlock[0].x, dimGrid[0].y = 1, dimGrid[0].z = 1;
        dimBlock[1].x = 8, dimBlock[1].y = 8, dimBlock[1].z = 1;
        dimGrid[1].x = width / dimBlock[1].x, dimGrid[1].y = height / dimBlock[1].y, dimGrid[1].z = 1;

        // Declare vars for host data and results
        struct ElementAttr job[N];
        struct ElementSet h_data[N], h_result[N];

        // Declare vars for device data and results
        struct ElementSet d_data[N], d_result[N];

        // Set job attributes
        job[0].type = dt_chr, job[0].dataSize = BYTES_INPUT, job[0].resultSize = dimGrid[0].x * dimBlock[0].x * MD5_LEN / sizeof(char);
        job[1].type = dt_flt, job[1].dataSize = 2 * height * width, job[1].resultSize = dimGrid[1].x * dimBlock[1].x;


        gettimeofday(&t1, NULL);
        // Allocate memory 
        for(int i = 0; i < n; i++){
            // printf("%s\n", "0-loop-allocateMem");
            switch(job[i].type){
                case dt_chr:
                    // printf("%s\n", "0-1");
                    CUDA_SAFE_CALL( hipHostMalloc((void**)&h_data[i].chr_data, sizeof(char) * job[i].dataSize) );      // host pinned
                    CUDA_SAFE_CALL( hipHostMalloc((void**)&h_result[i].chr_data, sizeof(char) * job[i].resultSize) );      // host pinned
                    // Data->chr_data = (unsigned char *)malloc(sizeof(char) * DataSize);
                    break;
                case dt_int:
                    // printf("%s\n", "0-2");
                    CUDA_SAFE_CALL( hipHostMalloc((void**)&h_data[i].int_data, sizeof(int) * job[i].dataSize) );      // host pinned
                    CUDA_SAFE_CALL( hipHostMalloc((void**)&h_result[i].int_data, sizeof(int) * job[i].resultSize) );      // host pinned
                    // Data->int_data = (int *)malloc(sizeof(int) * DataSize);
                    break;
                case dt_flt:
                    // printf("%s\n", "0-3");
                    CUDA_SAFE_CALL( hipHostMalloc((void**)&h_data[i].flt_data, sizeof(float) * job[i].dataSize) );      // host pinned
                    CUDA_SAFE_CALL( hipHostMalloc((void**)&h_result[i].flt_data, sizeof(float) * job[i].resultSize) );      // host pinned
                    // Data->flt_data = (float *)malloc(sizeof(float) * DataSize);
                    break;
            }
        }

        // init
        srand(2018);

        // initialize host data
        for (int i = 0; i < job[0].dataSize; i++)
            h_data[0].chr_data[i] = (unsigned char)(rand() % 256);

            // printf("%s\n", "0.5-init-1");

        randomInit(h_data[1].flt_data, job[1].dataSize);

        for(int i = 0; i < n; i++){
            // printf("%s\n", "1-loop-allocateDeviceMem");
                switch(job[i].type){
                case dt_chr:
                // printf("%s\n", "1-1");
                    CUDA_SAFE_CALL(hipMalloc((void **)&d_data[i].chr_data, sizeof(char) * job[i].dataSize));
                    CUDA_SAFE_CALL(hipMalloc((void **)&d_result[i].chr_data, sizeof(char) * job[i].resultSize));
                    break;
                case dt_int:
                // printf("%s\n", "1-2");
                    CUDA_SAFE_CALL(hipMalloc((void **)&d_data[i].int_data, sizeof(int) * job[i].dataSize));
                    CUDA_SAFE_CALL(hipMalloc((void **)&d_result[i].int_data, sizeof(int) * job[i].resultSize));
                    break;
                case dt_flt:
                // printf("%s\n", "1-3");
                    CUDA_SAFE_CALL(hipMalloc((void **)&d_data[i].flt_data, sizeof(float) * job[i].dataSize));
                    CUDA_SAFE_CALL(hipMalloc((void **)&d_result[i].flt_data, sizeof(float) * job[i].resultSize));
                    break;
            }
            // mallocMemoryOnDevice(&d_data[i], job[i].type, job[i].dataSize);
            // mallocMemoryOnDevice(&d_result[i], job[i].type, job[i].resultSize);
        }




	// read r and build hash table
	gettimeofday(&t1, NULL);

	CUDA_SAFE_CALL(hipHostMalloc((void**)&h_r, sizeof(record_t) * RLEN));

	if(read_r(h_r, &rlen)) {
		fprintf(stderr, "failed to read r\n");
		return -1;
	}

	gettimeofday(&t2, NULL);
	printf("Time on reading R: %lf ms\n", TIME_DIFF(t1, t2));

	gettimeofday(&t1, NULL);

	ht_r.n_buckets = RBUCKETS;
	// printf("Begin build_hash_table(r)\n");

	// varaibales for building hash table
	int build_hash_blocks = 64, build_hash_threads_per_block = 128;
	int *d_hist = NULL, *d_loc = NULL;
	record_t *d_r = NULL;
	int ret = 0;




	ht_r.d_rec = NULL;
	ht_r.d_idx = NULL;
	ht_r.n_records = rlen;
	if(!ht_r.n_buckets) {
		ht_r.n_buckets = NR_BUCKETS_DEFAULT;
	}

	// for scan
	int *d_sumbuf;	// the buffer used to store sum updates across subarrays
	int *h_sumbuf;
	int sum_tot, sum_delta;
	int scan_blocks = 512, scan_chunks;
	int scan_threads_per_block = 128;
	int scan_elems_per_block = 2 * scan_threads_per_block;
	int bytes_smem = sizeof(int) * scan_elems_per_block;



	// step 1: partition the array into many subarrays,
	// each of which is scanned separately
	scan_chunks = build_hash_blocks * build_hash_threads_per_block * ht_r.n_buckets / scan_elems_per_block;
	scan_chunks += (build_hash_blocks * build_hash_threads_per_block * ht_r.n_buckets % scan_elems_per_block) ? 1 : 0;


	// copy records to GPU device memory
	CUDA_SAFE_CALL(hipMalloc((void **)&d_r, rlen * sizeof(record_t)));

	// build histogram matrix to collect how many
    // records each thread generates in each bucket
	CUDA_SAFE_CALL(hipMalloc((void **)&d_hist,
	        sizeof(int) * build_hash_blocks * build_hash_threads_per_block * ht_r.n_buckets));

	// prefix sum to get the offsets
	CUDA_SAFE_CALL(hipMalloc((void **)&d_loc,
	        sizeof(int) * build_hash_blocks * build_hash_threads_per_block * ht_r.n_buckets));

	// build the hash table
	CUDA_SAFE_CALL(hipMalloc((void **)&ht_r.d_rec, rlen * sizeof(record_t)));


	CUDA_SAFE_CALL(hipMalloc((void **)&ht_r.d_idx, (ht_r.n_buckets + 1) * sizeof(int)));


		CUDA_SAFE_CALL(hipMalloc((void **)&d_sumbuf, sizeof(int) * scan_chunks));

		// printf("scan: begin hipHostMalloc\n");
		CUDA_SAFE_CALL(hipHostMalloc((void**)&h_sumbuf, sizeof(int) * scan_chunks));
		// printf("scan: finish hipHostMalloc\n");


    // printf("build_hash_table: begin hipMemcpyAsync(r)\n");
    CUDA_SAFE_CALL(hipMemcpyAsync(d_r, h_r, rlen * sizeof(record_t), hipMemcpyHostToDevice, stream[0]));


    	for (int i = 0; i < n; ++i) {
            // printf("%s\n", "2-loop-copyHtoD");
            switch(job[i].type){
                case dt_chr:
                    // printf("%s\n", "2-1-0");
                    // CUDA_SAFE_CALL(hipMemcpy(d_data[i].chr_data, h_data[i].chr_data, sizeof(char) * job[i].dataSize, hipMemcpyHostToDevice));
                    // printf("%s\n", "2-1");
                    CUDA_SAFE_CALL(hipMemcpyAsync(d_data[i].chr_data, h_data[i].chr_data, sizeof(char) * job[i].dataSize, hipMemcpyHostToDevice, stream[i+1]));
                    break;
                case dt_int:
                    // CUDA_SAFE_CALL(hipMemcpy(d_data[i].int_data, h_data[i].int_data, sizeof(int) * job[i].dataSize, hipMemcpyHostToDevice));
                    // printf("%s\n", "2-2");
                    CUDA_SAFE_CALL(hipMemcpyAsync(d_data[i].int_data, h_data[i].int_data, sizeof(int) * job[i].dataSize, hipMemcpyHostToDevice, stream[i+1]));
                    break;
                case dt_flt:
                    // CUDA_SAFE_CALL(hipMemcpy(d_data[i].flt_data, h_data[i].flt_data, sizeof(float) * job[i].dataSize, hipMemcpyHostToDevice));
                    // printf("%s\n", "2-3");
                    CUDA_SAFE_CALL(hipMemcpyAsync(d_data[i].flt_data, h_data[i].flt_data, sizeof(float) * job[i].dataSize, hipMemcpyHostToDevice, stream[i+1]));
                    break;
            }
        }


    // printf("build_hash_table: finish hipMemcpyAsync(r)\n");


	hash_build_hist<<<build_hash_blocks, build_hash_threads_per_block, 0, stream[0]>>>(d_hist, d_r, rlen,
	        ht_r.n_buckets);
	// printf("build_hash_table: finish hash_build_hist\n");
	if(hipStreamSynchronize(stream[0]) != hipSuccess) {
		fprintf(stderr, "kernel failed at hash_build_hist\n");
		ret = -1;
		goto failed;
	}

		for (int i = 0; i < n; ++i) {
            // printf("%s\n", "3-loop-execute-kernel");
            switch(i){
                case 0:
                    // printf("%s\n", "3-1");
                    md5_kernel<<<dimGrid[i], dimBlock[i], 0, stream[i+1]>>>(d_data[i].chr_data, d_result[i].chr_data, job[i].dataSize);
                    CUDA_SAFE_CALL(hipStreamSynchronize(stream[i+1]));
                    break;
                case 1:
                    // printf("%s\n", "3-2");
                    MatMulKernel<<<dimGrid[i], dimBlock[i], 0, stream[i+1]>>>(d_data[i].flt_data, d_result[i].flt_data, width);
                    break;
            }
        }

	

	// printf("build_hash_table: begin scan\n");


	// printf("scan: begin prefix_sum\n");
	prefix_sum<<<scan_blocks, scan_threads_per_block, bytes_smem, stream[0]>>>(
	        d_loc, d_sumbuf, d_hist, scan_chunks, build_hash_blocks * build_hash_threads_per_block * ht_r.n_buckets);
	// printf("scan: finish prefix_sum\n");
	// printf("scan: begin hipDeviceSynchronize\n");
	if(hipStreamSynchronize(stream[0]) != hipSuccess) {
		fprintf(stderr, "kernel failed at prefix_sum\n");
		goto failedScan;
	}

	// Copy result back to host
        for (int i = 0; i < n; ++i) {
            // printf("%s\n", "4-copy DtoH");
            switch(job[i].type){
                case dt_chr:
                    // CUDA_SAFE_CALL(hipMemcpy(h_result[i].chr_data, d_result[i].chr_data, sizeof(char) * job[i].resultSize, hipMemcpyDeviceToHost));
                    // printf("%s\n", "4-1");
                    CUDA_SAFE_CALL(hipMemcpyAsync(h_result[i].chr_data, d_result[i].chr_data, sizeof(char) * job[i].resultSize, hipMemcpyDeviceToHost, stream[i+1]));
                    break;
                case dt_int:
                    // CUDA_SAFE_CALL(hipMemcpy(h_result[i].int_data, d_result[i].int_data, sizeof(int) * job[i].resultSize, hipMemcpyDeviceToHost));
                    // printf("%s\n", "4-2");
                    CUDA_SAFE_CALL(hipMemcpyAsync(h_result[i].int_data, d_result[i].int_data, sizeof(int) * job[i].resultSize, hipMemcpyDeviceToHost, stream[i+1]));
                    break;
                case dt_flt:
                    // CUDA_SAFE_CALL(hipMemcpy(h_result[i].flt_data, d_result[i].flt_data, sizeof(float) * job[i].resultSize, hipMemcpyDeviceToHost));
                    // printf("%s\n", "4-3");
                    CUDA_SAFE_CALL(hipMemcpyAsync(h_result[i].flt_data, d_result[i].flt_data, sizeof(float) * job[i].resultSize, hipMemcpyDeviceToHost, stream[i+1]));
                    break;
            }
        }
	// printf("scan: finish hipDeviceSynchronize\n");

	// free(h_sumbuf);
	// hipFree(d_sumbuf);

	// step 2: update all scanned subarrays to derive the final result
	// res = hipMemcpy(h_sumbuf, d_sumbuf, sizeof(int) * nr_chunks,
	//         hipMemcpyDeviceToHost);

	// printf("scan: begin hipMemcpyAsync\n");
	CUDA_SAFE_CALL(hipMemcpyAsync(h_sumbuf, d_sumbuf, sizeof(int) * scan_chunks,
	        hipMemcpyDeviceToHost, stream[0]));
	// printf("scan: finish hipMemcpyAsync\n");

	sum_tot = 0;
	sum_delta = h_sumbuf[0];
	for(int i = 1; i < scan_chunks; i++) {
		sum_tot += sum_delta;
		sum_delta = h_sumbuf[i];
		h_sumbuf[i] = sum_tot;
	}
	h_sumbuf[0] = 0;
	sum_tot += sum_delta;

	// res = hipMemcpy(d_sumbuf, h_sumbuf, sizeof(int) * nr_chunks,
	//         hipMemcpyHostToDevice);
	// printf("scan: begin hipMemcpyAsync\n");
	CUDA_SAFE_CALL(hipMemcpyAsync(d_sumbuf, h_sumbuf, sizeof(int) * scan_chunks,
	        hipMemcpyHostToDevice, stream[0]));
	// printf("scan: finish hipMemcpyAsync\n");

	// printf("scan: begin prefix_sum_update\n");
	prefix_sum_update<<<scan_blocks, scan_threads_per_block, 0, stream[0]>>>(d_loc, d_sumbuf,
	        scan_chunks, build_hash_blocks * build_hash_threads_per_block * ht_r.n_buckets);
	// printf("scan: finish prefix_sum_update\n");
	// printf("scan: begin hipDeviceSynchronize\n");
	if(hipStreamSynchronize(stream[0]) != hipSuccess) {
		fprintf(stderr, "kernel failed at prefix_sum_update\n");
		goto failedScan;
	}
	// printf("scan: finish hipDeviceSynchronize\n");

	goto finishScan;

failedScan:
	// printf("scan: free\n");
	hipFree(h_sumbuf);
	// printf("scan: cudafree\n");
	hipFree(d_sumbuf);


finishScan:
	// printf("scan: free\n");
	hipFree(h_sumbuf);
	// printf("scan: cudafree\n");
	hipFree(d_sumbuf);


// printf("build_hash_table: finish scan\n");
	CUDA_FREE(d_hist);



	hash_build<<<build_hash_blocks, build_hash_threads_per_block, 0, stream[0]>>>(ht_r.d_rec, ht_r.d_idx,
	        d_r, rlen, d_loc, ht_r.n_buckets);
	if(hipStreamSynchronize(stream[0]) != hipSuccess) {
		fprintf(stderr, "kernel failed at hash_build\n");
		ret = -1;
		goto failed;
	}

	goto finish;

failed:
	free_hash_table(&ht_r);

finish:
	CUDA_FREE(d_r);
	CUDA_FREE(d_hist);
	CUDA_FREE(d_loc);


	hipFree(h_r);	// table R on the host is not needed any more

	gettimeofday(&t2, NULL);
	printf("Time on building hash table for R: %lf ms\n", TIME_DIFF(t1, t2));


	// for each chunk of s, join with r
	// h_s = (record_t *)malloc(sizeof(record_t) * S_CHUNK_LEN);
	// if(!h_s) {
	// 	fprintf(stderr, "malloc failed for s\n");
	// 	free_hash_table(&ht_r);
	// 	return -1;
	// }

	gettimeofday(&t1, NULL);
	for(int i = 0; i < S_CHUNK_CNT; i++){
		CUDA_SAFE_CALL(hipHostMalloc((void**)&h_s[i], sizeof(record_t) * S_CHUNK_LEN));
		slen = read_s(h_s[i], S_CHUNK_LEN, 0);
	}
	gettimeofday(&t2, NULL);
	printf("Time on reading S: %lf ms ( %lf ms per join )\n", TIME_DIFF(t1, t2), TIME_DIFF(t1, t2)/S_CHUNK_CNT);


	// The number of result records joined per chunk is approximately:
	// RLEN * S_CHUNK_LEN / max(RKEY_MAX, SKEY_MAX)
	gettimeofday(&t1, NULL);
	for(int i = 0; i < S_CHUNK_CNT; i++) {

		// printf("%d\n", i);
		// join with r
		if(slen > 0) {
			// printf("Begin hash_join\n");
			if(hash_join(NULL, NULL, &ht_r, h_s[i], slen, stream, i)) {
				fprintf(stderr, "hash join failed for the %dth chunk of S\n",
				        i);
				break;
			}
			// printf("Finish hash_join\n");
		}
		else {
			fprintf(stderr, "failed to read s\n");
			break;
		}
	}
	gettimeofday(&t2, NULL);
	printf("Time on hash join: %lf ms ( %lf ms per join )\n", TIME_DIFF(t1, t2), TIME_DIFF(t1, t2)/S_CHUNK_CNT);

	free_hash_table(&ht_r);
	hipFree(h_s);

	gettimeofday(&t_end, NULL);
	printf("Total time taken: %lf ms\n", TIME_DIFF(t_start, t_end));
	printf("Time of ending hj: %lf \n", TVAL(t_end));
	return 0;
}
