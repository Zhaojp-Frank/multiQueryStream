#include <math.h>
#include <stdio.h>
#include <stdlib.h>

#include "hj.h"
#include "hj_kernels.cu"

// default number of buckets chosen by build_hash_table
#define NR_BUCKETS_DEFAULT	256

// prefix sum. the sum of all elements in d_src is returned
// if successful; otherwise, -1 is returned.
int scan(int *d_dest, int *d_src, int len, hipStream_t* Stream, int StreamID)
{
	hipError_t res;
	int *d_sumbuf;	// the buffer used to store sum updates across subarrays
	int *h_sumbuf;
	int sum_tot, sum_delta;
	int nr_blocks = 512, nr_chunks;
	int nr_threads_per_block = 128;
	int nr_elems_per_block = 2 * nr_threads_per_block;
	int bytes_smem = sizeof(int) * nr_elems_per_block;

	// step 1: partition the array into many subarrays,
	// each of which is scanned separately
	nr_chunks = len / nr_elems_per_block;
	nr_chunks += (len % nr_elems_per_block) ? 1 : 0;

	res = hipMalloc((void **)&d_sumbuf, sizeof(int) * nr_chunks);
	if(res != hipSuccess) {
		fprintf(stderr, "cudaMemAlloc(&d_sumbuf) failed\n");
		return -1;
	}

	// h_sumbuf = (int *)malloc(sizeof(int) * nr_chunks);
	// if(!h_sumbuf) {
	// 	fprintf(stderr, "malloc() failed for h_sumbuf\n");
	// 	hipFree(d_sumbuf);
	// 	return -1;
	// }

	// printf("scan: begin hipHostMalloc\n");
	res = hipHostMalloc((void**)&h_sumbuf, sizeof(int) * nr_chunks);
	if(res != hipSuccess) {
		fprintf(stderr, "hipHostMalloc(&h_sumbuf) failed\n");
		return -1;
	}
	// printf("scan: finish hipHostMalloc\n");

	// printf("scan: begin prefix_sum\n");
	prefix_sum<<<nr_blocks, nr_threads_per_block, bytes_smem, Stream[StreamID]>>>(
	        d_dest, d_sumbuf, d_src, nr_chunks, len);
	// printf("scan: finish prefix_sum\n");
	// printf("scan: begin hipDeviceSynchronize\n");
	if(hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "kernel failed at prefix_sum\n");
		free(h_sumbuf);
		hipFree(d_sumbuf);
		return -1;
	}
	// printf("scan: finish hipDeviceSynchronize\n");

	// free(h_sumbuf);
	// hipFree(d_sumbuf);

	// step 2: update all scanned subarrays to derive the final result
	// res = hipMemcpy(h_sumbuf, d_sumbuf, sizeof(int) * nr_chunks,
	//         hipMemcpyDeviceToHost);

	// printf("scan: begin hipMemcpyAsync\n");
	res = hipMemcpyAsync(h_sumbuf, d_sumbuf, sizeof(int) * nr_chunks,
	        hipMemcpyDeviceToHost, Stream[StreamID]);
	// printf("scan: finish hipMemcpyAsync\n");
	if(res != hipSuccess) {
		fprintf(stderr, "hipMemcpy DtoH failed\n");
		free(h_sumbuf);
		hipFree(d_sumbuf);
		return -1;
	}

	sum_tot = 0;
	sum_delta = h_sumbuf[0];
	for(int i = 1; i < nr_chunks; i++) {
		sum_tot += sum_delta;
		sum_delta = h_sumbuf[i];
		h_sumbuf[i] = sum_tot;
	}
	h_sumbuf[0] = 0;
	sum_tot += sum_delta;

	// res = hipMemcpy(d_sumbuf, h_sumbuf, sizeof(int) * nr_chunks,
	//         hipMemcpyHostToDevice);
	// printf("scan: begin hipMemcpyAsync\n");
	res = hipMemcpyAsync(d_sumbuf, h_sumbuf, sizeof(int) * nr_chunks,
	        hipMemcpyHostToDevice, Stream[StreamID]);
	// printf("scan: finish hipMemcpyAsync\n");
	if(res != hipSuccess) {
		fprintf(stderr, "hipMemcpy HtoD failed\n");
		free(h_sumbuf);
		hipFree(d_sumbuf);
		return -1;
	}

	// printf("scan: begin prefix_sum_update\n");
	prefix_sum_update<<<nr_blocks, nr_threads_per_block, 0, Stream[StreamID]>>>(d_dest, d_sumbuf,
	        nr_chunks, len);
	// printf("scan: finish prefix_sum_update\n");
	// printf("scan: begin hipDeviceSynchronize\n");
	if(hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "kernel failed at prefix_sum_update\n");
		free(h_sumbuf);
		hipFree(d_sumbuf);
		return -1;
	}
	// printf("scan: finish hipDeviceSynchronize\n");

	// printf("scan: free\n");
	hipFree(h_sumbuf);
	// printf("scan: cudafree\n");
	hipFree(d_sumbuf);
	return sum_tot;
}

int build_hash_table(hash_table_t *ht, record_t *h_r, int rlen, hipStream_t* Stream, int StreamID)
{
	int nr_blocks = 64, nr_threads_per_block = 128;
	int *d_hist = NULL, *d_loc = NULL;
	record_t *d_r = NULL;
	hipError_t res;
	int ret = 0;

	ht->d_rec = NULL;
	ht->d_idx = NULL;
	ht->n_records = rlen;
	if(!ht->n_buckets) {
		ht->n_buckets = NR_BUCKETS_DEFAULT;
	}

	// copy records to GPU device memory
	res = hipMalloc((void **)&d_r, rlen * sizeof(record_t));
	if(res != hipSuccess) {
		fprintf(stderr, "hipMalloc(&d_r) failed\n");
		ret = -1;
		goto failed;
	}

    // res = hipMemcpy(d_r, h_r, rlen * sizeof(record_t), hipMemcpyHostToDevice);
    // printf("build_hash_table: begin hipMemcpyAsync(r)\n");
    res = hipMemcpyAsync(d_r, h_r, rlen * sizeof(record_t), hipMemcpyHostToDevice, Stream[StreamID]);
    if(res != hipSuccess) {
        fprintf(stderr, "hipMemcpy(r) failed\n");
 		ret = -1;
		goto failed;
    }

    // printf("build_hash_table: finish hipMemcpyAsync(r)\n");

	// build histogram matrix to collect how many
    // records each thread generates in each bucket
	res = hipMalloc((void **)&d_hist,
	        sizeof(int) * nr_blocks * nr_threads_per_block * ht->n_buckets);
	if(res != hipSuccess) {
		fprintf(stderr, "hipMalloc(&d_hist) failed\n");
		ret = -1;
		goto failed;
	}

	hash_build_hist<<<nr_blocks, nr_threads_per_block, 0, Stream[StreamID]>>>(d_hist, d_r, rlen,
	        ht->n_buckets);
	// printf("build_hash_table: finish hash_build_hist\n");
	if(hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "kernel failed at hash_build_hist\n");
		ret = -1;
		goto failed;
	}

	// prefix sum to get the offsets
	res = hipMalloc((void **)&d_loc,
	        sizeof(int) * nr_blocks * nr_threads_per_block * ht->n_buckets);
	if(res != hipSuccess) {
		fprintf(stderr, "hipMalloc(&d_loc) failed\n");
		ret = -1;
		goto failed;
	}

	// printf("build_hash_table: begin scan\n");
	if(scan(d_loc, d_hist, nr_blocks * nr_threads_per_block * ht->n_buckets, Stream, StreamID)
	        < 0) {
		fprintf(stderr, "scan failed\n");
		ret = -1;
		goto failed;
	}
	// printf("build_hash_table: finish scan\n");
	CUDA_FREE(d_hist);
	d_hist = NULL;

	// build the hash table
	res = hipMalloc((void **)&ht->d_rec, rlen * sizeof(record_t));
	if(res != hipSuccess) {
		fprintf(stderr, "hipMalloc(&ht->d_rec) failed\n");
		ret = -1;
		goto failed;
	}

	res = hipMalloc((void **)&ht->d_idx, (ht->n_buckets + 1) * sizeof(int));
	if(res != hipSuccess) {
		fprintf(stderr, "hipMalloc(&ht->d_idx) failed\n");
		ret = -1;
		goto failed;
	}


	hash_build<<<nr_blocks, nr_threads_per_block, 0, Stream[StreamID]>>>(ht->d_rec, ht->d_idx,
	        d_r, rlen, d_loc, ht->n_buckets);
	if(hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "kernel failed at hash_build\n");
		ret = -1;
		goto failed;
	}

	goto finish;

failed:
	free_hash_table(ht);

finish:
	CUDA_FREE(d_r);
	CUDA_FREE(d_hist);
	CUDA_FREE(d_loc);
	return ret;
}

void free_hash_table(hash_table_t *ht)
{
	CUDA_FREE(ht->d_rec);
	CUDA_FREE(ht->d_idx);
	ht->d_rec = NULL;
	ht->d_idx = NULL;
	ht->n_records = 0;
	ht->n_buckets = 0;
}

int hash_join(record_t **h_res, int *reslen,
        hash_table_t *ht_r, record_t *h_s, int slen, hipStream_t* Stream, int StreamID)
{
	hipError_t res;
	int ret = 0, n_results;
	record_t *restmp = NULL;
	int nr_blocks = 256, nr_threads_per_block = 128;
	int *d_hist = NULL, *d_loc = NULL;
	record_t *d_s = NULL, *d_res = NULL;

	// copy S to GPU device memory
	res = hipMalloc((void **)&d_s, slen * sizeof(record_t));
	if(res != hipSuccess) {
		fprintf(stderr, "hipMalloc(&d_s) failed\n");
		ret = -1;
		goto failed;
	}

	// printf("hash_join: begin hipMemcpyAsync\n");
    // res = hipMemcpy(d_s, h_s, slen * sizeof(record_t), hipMemcpyHostToDevice);
    res = hipMemcpyAsync(d_s, h_s, slen * sizeof(record_t), hipMemcpyHostToDevice, Stream[StreamID]);
    // printf("hash_join: finish hipMemcpyAsync\n");
    if(res != hipSuccess) {
        fprintf(stderr, "hipMemcpyHtoD(s) failed\n");
 		ret = -1;
       goto failed;
    }

	// count the number of records joined by each thread
	res = hipMalloc((void **)&d_hist,
	        sizeof(int) * nr_blocks * nr_threads_per_block);
	if(res != hipSuccess) {
		fprintf(stderr, "hipMalloc(&d_hist) failed\n");
		ret = -1;
		goto failed;
	}

	// printf("hash_join: begin hash_join_hist\n");
	hash_join_hist<<<nr_blocks, nr_threads_per_block, 0, Stream[StreamID]>>>(d_hist, ht_r->d_rec,
	        ht_r->d_idx, ht_r->n_buckets, d_s, slen);
	// printf("hash_join: finish hash_join_hist\n");
	if(hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "kernel failed at hash_join_hist\n");
		ret = -1;
		goto failed;
	}

	// prefix sum to get the locations
	res = hipMalloc((void **)&d_loc,
	        sizeof(int) * nr_blocks * nr_threads_per_block);
	if(res != hipSuccess) {
		fprintf(stderr, "hipMalloc(&d_loc) failed\n");
		ret = -1;
		goto failed;
	}

	// printf("hash_join: begin scan\n");
	n_results = scan(d_loc, d_hist, nr_blocks * nr_threads_per_block, Stream, StreamID);
	if(n_results < 0) {
		fprintf(stderr, "scan failed\n");
		ret = -1;
		goto failed;
	}
	// printf("hash_join: finish scan\n");
	CUDA_FREE(d_hist);
	d_hist = NULL;

	if(n_results <= 0) {
		if(h_res) {
			*h_res = NULL;
		}

		if(reslen) {
			*reslen = 0;
		}

		goto finish;
	}

	// do hash join
	res = hipMalloc((void **)&d_res, n_results * sizeof(record_t));
	if(res != hipSuccess) {
		fprintf(stderr, "hipMalloc(&d_res) failed\n");
		ret = -1;
		goto failed;
	}

	// printf("hash_join: begin hash_join\n");
	hash_join<<<nr_blocks, nr_threads_per_block, 0, Stream[StreamID]>>>(d_res, d_loc, ht_r->d_rec,
	        ht_r->d_idx, ht_r->n_buckets, d_s, slen);
	// printf("hash_join: finish hash_join\n");
	if(hipDeviceSynchronize() != hipSuccess) {
		fprintf(stderr, "kernel failed at hash_join\n");
		ret = -1;
		goto failed;
	}

	// setting return values, if required
	if(h_res) {
		// restmp = (record_t *)malloc(n_results * sizeof(record_t));
		// if(!restmp) {
		// 	fprintf(stderr, "malloc failed for h_res\n");
		// 	ret = -1;
		// 	goto failed;
		// }
		res = hipHostMalloc((void**)&restmp, n_results * sizeof(record_t));
		if(res != hipSuccess) {
			fprintf(stderr, "hipHostMalloc(&h_res) failed\n");
			ret = -1;
			goto failed;
		}

		// res = hipMemcpy((void *)restmp, d_res, n_results * sizeof(record_t),
		//         hipMemcpyDeviceToHost);
		// printf("hash_join: begin hipMemcpyAsync\n");
		res = hipMemcpyAsync((void *)restmp, d_res, n_results * sizeof(record_t),
		        hipMemcpyDeviceToHost, Stream[StreamID]);
		// printf("hash_join: finish hipMemcpyAsync\n");
		if (res != hipSuccess) {
			printf("cudaMemcpyDtoH failed when getting join results\n");
			ret = -1;
			goto failed;
		}
		*h_res = restmp;
	}

	if(reslen) {
		*reslen = n_results;
	}

	goto finish;

failed:
	hipFree(restmp);

finish:
	CUDA_FREE(d_s);
	CUDA_FREE(d_hist);
	CUDA_FREE(d_loc);
	CUDA_FREE(d_res);
	return ret;
}
