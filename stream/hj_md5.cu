#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>

#ifdef USE_MQX
#include "mqx.h"
#endif

#include "../md5/defs.h"
#include "../md5/kernels.cu"

// Memory object sizes:
// 1. hash table build: 2 * 8 * RLEN + 2 * 32 * 1024 * RBUCKETS
// 2. after hash_build before hash_join: 8 * RLEN
// 3. each hash_join: 8 * S_CHUNK_LEN + 8 * RLEN + 8 * n_results

#include "hj.h"

// number of records in R
//#define RLEN      (40 * 1024L * 1024L)
#define RLEN        (10L * 1024L * 1024L)
// max of R's keys
#define RKEY_MAX    (1024 * 256)
// seed of R's keys
#define RKEY_SEED   1
// number of buckets for R's hash table; should not be larger than RKEY_MAX
#define RBUCKETS    (1024 * 8) // must be power of 2

// max of S's keys
#define SKEY_MAX    (1024 * 256)
// seed of S's keys
#define SKEY_SEED   2

// number of records in each chunk read from S
#define S_CHUNK_LEN     (64L * 1024L)
// how many chunks to be read from S
#define S_CHUNK_CNT     5

int read_r(record_t **r, int *rlen)
{
    size_t size_r = sizeof(record_t) * RLEN;

    record_t *r_tmp = (record_t *)malloc(size_r);
    if(!r_tmp) {
        fprintf(stderr, "malloc failed for R\n");
        return -1;
    }

    unsigned int seed = RKEY_SEED;
    for(int i = 0; i < RLEN; i++) {
        r_tmp[i].y = rand_r(&seed) % RKEY_MAX;
        r_tmp[i].x = i;
    }

    *r = r_tmp;
    *rlen = RLEN;
    return 0;
}

// return the number of records actually read
int read_s(record_t *s, int slen, int skey_start)
{
    CUDA_SAFE_CALL( hipHostMalloc((void**)&s, sizeof(int) * slen * 2) );      // host pinned
    static unsigned int seed = SKEY_SEED;
    for(int i = 0; i < slen; i++) {
        s[i].y = rand_r(&seed) % (SKEY_MAX - skey_start) + skey_start;
        s[i].x = skey_start + i;
    }
    return slen;
}


#define TVAL(t)         ((t).tv_sec * 1000.0 + (t).tv_usec / 1000.0)
#define TDIFF(t1, t2)   (TVAL(t2) - TVAL(t1))

#ifndef CUDA_SAFE_CALL
#define CUDA_SAFE_CALL(call) \
    do { \
        hipError_t err = call; \
        if(hipSuccess != err) { \
            fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \
                    __FILE__, __LINE__, hipGetErrorString(err)); \
            exit(EXIT_FAILURE); \
        } \
    } while (0)
#endif

void save_data_to_disk(unsigned char *data, size_t size)
{
    FILE *fout = fopen("md5.output", "w");
    if (!fout) {
        perror("Failed to create output file");
        exit(1);
    }
    fwrite(data, sizeof(unsigned char), size, fout);
    fclose(fout);
}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}


//float multiplication kernel called by MatMul()
__global__ void MatMulKernel(float *A, float *C, int Width)
{
        // Each thread computes one element of C by accumulating results into Cvalue
        float Cvalue = 0;
        int row = blockIdx.y * blockDim.y + threadIdx.y;
        int col = blockIdx.x * blockDim.x + threadIdx.x;

        for (int e = 0; e < Width; ++e)
                Cvalue += A[row * Width + e] * A[Width * Width + e * Width + col - 1];
        C[row * Width + col] = Cvalue;
}

#define N 3

enum DataType { dt_chr, dt_int, dt_flt };

struct ElementAttr{
    enum DataType type;
    int dataSize;
    int resultSize;
};

struct ElementSet {
  union {
    unsigned char *chr_data;
    int           *int_data;
    float         *flt_data;   
  };
};

int mallocMemory(struct ElementSet *Data, enum DataType Type, int DataSize){
    switch(Type){
        case dt_chr:
            CUDA_SAFE_CALL( hipHostMalloc((void**)&Data->chr_data, sizeof(char) * DataSize) );      // host pinned
            // Data->chr_data = (unsigned char *)malloc(sizeof(char) * DataSize);
            break;
        case dt_int:
            CUDA_SAFE_CALL( hipHostMalloc((void**)&Data->int_data, sizeof(int) * DataSize) );      // host pinned
            // Data->int_data = (int *)malloc(sizeof(int) * DataSize);
            break;
        case dt_flt:
            CUDA_SAFE_CALL( hipHostMalloc((void**)&Data->flt_data, sizeof(float) * DataSize) );      // host pinned
            // Data->flt_data = (float *)malloc(sizeof(float) * DataSize);
            break;
    }
    return 0;
}

int mallocMemoryOnDevice(struct ElementSet *Data, enum DataType Type, int DataSize){
    switch(Type){
        case dt_chr:
        printf("%s\n", "1-1");
            CUDA_SAFE_CALL(hipMalloc((void **)Data->chr_data, sizeof(char) * DataSize));
            break;
        case dt_int:
        printf("%s\n", "1-2");
            CUDA_SAFE_CALL(hipMalloc((void **)Data->int_data, sizeof(int) * DataSize));
            break;
        case dt_flt:
        printf("%s\n", "1-3");
            CUDA_SAFE_CALL(hipMalloc((void **)Data->flt_data, sizeof(float) * DataSize));
            break;
    }
    return 0;
}

int printElement(struct ElementSet Data, struct ElementAttr Job){
    switch(Job.type){
        case dt_chr:
            for (int j = 0; j < Job.dataSize; ++j)
                printf("%c\t", Data.chr_data[j]);
            printf("\n");
                break;
        case dt_int:
            for (int j = 0; j < Job.dataSize; ++j)
                printf("%d\t", Data.int_data[j]);
            printf("\n");
            break;
        case dt_flt:
            for (int j = 0; j < Job.dataSize; ++j)
                printf("%f\t", Data.flt_data[j]);
            printf("\n");
            break;
    }
    return 0;
}

int main()
{
    int n = N - 1;
    hipStream_t stream[N];

    struct timeval t1, t2;
    float pinned_time = 0;

    // int mm_block = 128;
    // int width = 16 * 128;
    // int height = width;

    int mm_block = 8;
    int width = 8 * 256;
    int height = width;

    // thread per block and block per grid for job n
    dim3 dimBlock[N],dimGrid[N];

    // dimBlock[0].x = 64, dimBlock[0].y = 1, dimBlock[0].z = 1;
    // dimGrid[0].x = 4096 / dimBlock[0].x, dimGrid[0].y = 1, dimGrid[0].z = 1;
    //md5
    dimBlock[0].x = 32, dimBlock[0].y = 1, dimBlock[0].z = 1;
    dimGrid[0].x = 4096 / dimBlock[0].x, dimGrid[0].y = 1, dimGrid[0].z = 1;
    //mm
    dimBlock[1].x = 8, dimBlock[1].y = 8, dimBlock[1].z = 1;
    dimGrid[1].x = width / dimBlock[1].x, dimGrid[1].y = height / dimBlock[1].y, dimGrid[1].z = 1;
    //hj
    dimBlock[1].x = 512, dimBlock[1].y = 1, dimBlock[1].z = 1;
    dimGrid[1].x = 8192 / dimBlock[1].x, dimGrid[1].y = 1, dimGrid[1].z = 1;

    // Declare vars for host data and results
    struct ElementAttr job[N];
    struct ElementSet h_data[N], h_result[N];

    // Declare vars for device data and results
    struct ElementSet d_data[N], d_result[N];

    // for hash
    record_t *h_r, *h_s;
    hash_table_t ht_r;
    int rlen, slen;


    // gettimeofday(&t1, NULL);
    ht_r.n_buckets = RBUCKETS;

    int *d_hist = NULL, *d_loc = NULL;
    record_t *d_r = NULL;
    hipError_t res;
    int ret = 0;

    ht_r->d_rec = NULL;
    ht_r->d_idx = NULL;
    ht_r->n_records = rlen;
    if(!ht_r->n_buckets) {
        ht_r->n_buckets = NR_BUCKETS_DEFAULT;
    }

    // if(build_hash_table(&ht_r, h_r, rlen)) {
    //     fprintf(stderr, "failed to build hash table for R\n");
    //     free(h_r);
    //     return -1;
    // }
    // gettimeofday(&t2, NULL);
    // printf("Time on building hash table for R: %lf ms\n", TIME_DIFF(t1, t2));

    // Set job attributes
    job[0].type = dt_chr, job[0].dataSize = BYTES_INPUT, job[0].resultSize = dimGrid[0].x * dimBlock[0].x * MD5_LEN / sizeof(char);
    job[1].type = dt_flt, job[1].dataSize = 2 * height * width, job[1].resultSize = dimGrid[1].x * dimBlock[1].x;


    gettimeofday(&t1, NULL);
    // Allocate memory 
    for(int i = 0; i < n; i++){
        // printf("%s\n", "0-loop-allocateMem");
        switch(job[i].type){
            case dt_chr:
                // printf("%s\n", "0-1");
                CUDA_SAFE_CALL( hipHostMalloc((void**)&h_data[i].chr_data, sizeof(char) * job[i].dataSize) );      // host pinned
                CUDA_SAFE_CALL( hipHostMalloc((void**)&h_result[i].chr_data, sizeof(char) * job[i].resultSize) );      // host pinned
                // Data->chr_data = (unsigned char *)malloc(sizeof(char) * DataSize);
                break;
            case dt_int:
                // printf("%s\n", "0-2");
                CUDA_SAFE_CALL( hipHostMalloc((void**)&h_data[i].int_data, sizeof(int) * job[i].dataSize) );      // host pinned
                CUDA_SAFE_CALL( hipHostMalloc((void**)&h_result[i].int_data, sizeof(int) * job[i].resultSize) );      // host pinned
                // Data->int_data = (int *)malloc(sizeof(int) * DataSize);
                break;
            case dt_flt:
                // printf("%s\n", "0-3");
                CUDA_SAFE_CALL( hipHostMalloc((void**)&h_data[i].flt_data, sizeof(float) * job[i].dataSize) );      // host pinned
                CUDA_SAFE_CALL( hipHostMalloc((void**)&h_result[i].flt_data, sizeof(float) * job[i].resultSize) );      // host pinned
                // Data->flt_data = (float *)malloc(sizeof(float) * DataSize);
                break;
        }

        // mallocMemory(&h_data[i], job[i].type, job[i].dataSize);
        // mallocMemory(&h_result[i], job[i].type, job[i].resultSize);
    }

    // hash 
    if(read_r(&h_r, &rlen)) {
        fprintf(stderr, "failed to read r\n");
        return -1;
    }

    gettimeofday(&t2, NULL);
    pinned_time = TDIFF(t1, t2);
    printf("pinned_time: %f ms\n", pinned_time);


    // printf("%s\n", "0-end loop");

    // init
    srand(2018);

    // printf("%s\n", "0.5-init");

    // initialize host data
    for (int i = 0; i < job[0].dataSize; i++)
        h_data[0].chr_data[i] = (unsigned char)(rand() % 256);

        // printf("%s\n", "0.5-init-1");
    
    // for (int i = 0; i < job[1].dataSize; ++i)
    //     h_data[1].flt_data[i] = rand() / (float)RAND_MAX;

    randomInit(h_data[1].flt_data, job[1].dataSize);

        // printf("%s\n", "0.5-init-2");

    // begin timing
    gettimeofday(&t1, NULL);

    // Allocate memory 
    for(int i = 0; i < n; i++){
        // printf("%s\n", "1-loop-allocateDeviceMem");
            switch(job[i].type){
            case dt_chr:
            // printf("%s\n", "1-1");
                CUDA_SAFE_CALL(hipMalloc((void **)&d_data[i].chr_data, sizeof(char) * job[i].dataSize));
                CUDA_SAFE_CALL(hipMalloc((void **)&d_result[i].chr_data, sizeof(char) * job[i].resultSize));
                break;
            case dt_int:
            // printf("%s\n", "1-2");
                CUDA_SAFE_CALL(hipMalloc((void **)&d_data[i].int_data, sizeof(int) * job[i].dataSize));
                CUDA_SAFE_CALL(hipMalloc((void **)&d_result[i].int_data, sizeof(int) * job[i].resultSize));
                break;
            case dt_flt:
            // printf("%s\n", "1-3");
                CUDA_SAFE_CALL(hipMalloc((void **)&d_data[i].flt_data, sizeof(float) * job[i].dataSize));
                CUDA_SAFE_CALL(hipMalloc((void **)&d_result[i].flt_data, sizeof(float) * job[i].resultSize));
                break;
        }
        // mallocMemoryOnDevice(&d_data[i], job[i].type, job[i].dataSize);
        // mallocMemoryOnDevice(&d_result[i], job[i].type, job[i].resultSize);
    }

    CUDA_SAFE_CALL(hipMalloc((void **)&d_r, rlen * sizeof(record_t)));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_hist, sizeof(int) * nr_blocks * nr_threads_per_block * ht_r->n_buckets));
    CUDA_SAFE_CALL(hipMalloc((void **)&d_loc, sizeof(int) * nr_blocks * nr_threads_per_block * ht_r->n_buckets));

    // printf("%s\n", "1-end-loop");

    // Create cuda stream
    for (int i = 0; i < N; ++i)
        CUDA_SAFE_CALL( hipStreamCreate(&stream[i]) );

    // Copy data from host to device
    for (int i = 0; i < n; ++i) {
        // printf("%s\n", "2-loop-copyHtoD");
        switch(job[i].type){
            case dt_chr:
                // printf("%s\n", "2-1-0");
                // CUDA_SAFE_CALL(hipMemcpy(d_data[i].chr_data, h_data[i].chr_data, sizeof(char) * job[i].dataSize, hipMemcpyHostToDevice));
                // printf("%s\n", "2-1");
                CUDA_SAFE_CALL(hipMemcpyAsync(d_data[i].chr_data, h_data[i].chr_data, sizeof(char) * job[i].dataSize, hipMemcpyHostToDevice, stream[i]));
                break;
            case dt_int:
                // CUDA_SAFE_CALL(hipMemcpy(d_data[i].int_data, h_data[i].int_data, sizeof(int) * job[i].dataSize, hipMemcpyHostToDevice));
                // printf("%s\n", "2-2");
                CUDA_SAFE_CALL(hipMemcpyAsync(d_data[i].int_data, h_data[i].int_data, sizeof(int) * job[i].dataSize, hipMemcpyHostToDevice, stream[i]));
                break;
            case dt_flt:
                // CUDA_SAFE_CALL(hipMemcpy(d_data[i].flt_data, h_data[i].flt_data, sizeof(float) * job[i].dataSize, hipMemcpyHostToDevice));
                // printf("%s\n", "2-3");
                CUDA_SAFE_CALL(hipMemcpyAsync(d_data[i].flt_data, h_data[i].flt_data, sizeof(float) * job[i].dataSize, hipMemcpyHostToDevice, stream[i]));
                break;
        }
    }

    CUDA_SAFE_CALL(hipMemcpyAsync(d_r, h_r, rlen * sizeof(record_t), hipMemcpyHostToDevice, stream[2]));
    // printf("%s\n", "2-end-loop");

    for (int i = 0; i < n; ++i) {
        // printf("%s\n", "3-loop-execute-kernel");
        switch(i){
            case 0:
                // printf("%s\n", "3-1");
                md5_kernel<<<dimGrid[i], dimBlock[i], 0, stream[i]>>>(d_data[i].chr_data, d_result[i].chr_data, job[i].dataSize);
                CUDA_SAFE_CALL(hipDeviceSynchronize());
                break;
            case 1:
                // printf("%s\n", "3-2");
                MatMulKernel<<<dimGrid[i], dimBlock[i], 0, stream[i]>>>(d_data[i].flt_data, d_result[i].flt_data, width);
                break;
        }
    }

    hash_build_hist<<<nr_blocks, nr_threads_per_block, 0, stream[3]>>>(d_hist, d_r, rlen, ht_r->n_buckets);
    // printf("%s\n", "3-end loop");

    // Copy result back to host
    for (int i = 0; i < n; ++i) {
        // printf("%s\n", "4-copy DtoH");
        switch(job[i].type){
            case dt_chr:
                // CUDA_SAFE_CALL(hipMemcpy(h_result[i].chr_data, d_result[i].chr_data, sizeof(char) * job[i].resultSize, hipMemcpyDeviceToHost));
                // printf("%s\n", "4-1");
                CUDA_SAFE_CALL(hipMemcpyAsync(h_result[i].chr_data, d_result[i].chr_data, sizeof(char) * job[i].resultSize, hipMemcpyDeviceToHost, stream[i]));
                break;
            case dt_int:
                // CUDA_SAFE_CALL(hipMemcpy(h_result[i].int_data, d_result[i].int_data, sizeof(int) * job[i].resultSize, hipMemcpyDeviceToHost));
                // printf("%s\n", "4-2");
                CUDA_SAFE_CALL(hipMemcpyAsync(h_result[i].int_data, d_result[i].int_data, sizeof(int) * job[i].resultSize, hipMemcpyDeviceToHost, stream[i]));
                break;
            case dt_flt:
                // CUDA_SAFE_CALL(hipMemcpy(h_result[i].flt_data, d_result[i].flt_data, sizeof(float) * job[i].resultSize, hipMemcpyDeviceToHost));
                // printf("%s\n", "4-3");
                CUDA_SAFE_CALL(hipMemcpyAsync(h_result[i].flt_data, d_result[i].flt_data, sizeof(float) * job[i].resultSize, hipMemcpyDeviceToHost, stream[i]));
                break;
        }
    }
    // printf("%s\n", "4-end loop");

    gettimeofday(&t2, NULL);
    printf("Computing took %f ms\n", TDIFF(t1, t2));

    for (int i = 0; i < n; ++i) {
        printf("%s\n", "5-cuda free");
        switch(job[i].type){
            case dt_chr:
                printf("%s\n", "5-1");
                CUDA_SAFE_CALL(hipFree(d_data[i].chr_data));
                printf("%s\n", "5-1-1");
                CUDA_SAFE_CALL(hipFree(d_result[i].chr_data));
                break;
            case dt_int:
                printf("%s\n", "5-2");
                CUDA_SAFE_CALL(hipFree(d_data[i].int_data));
                CUDA_SAFE_CALL(hipFree(d_result[i].int_data));
                break;
            case dt_flt:
                printf("%s\n", "5-3");
                CUDA_SAFE_CALL(hipFree(d_data[i].flt_data));
                CUDA_SAFE_CALL(hipFree(d_result[i].flt_data));
                break;
        }
    }
    printf("%s\n", "5-end loop");

    for (int i = 0; i < N; ++i)
        CUDA_SAFE_CALL( hipStreamDestroy(stream[i]) );

    // gettimeofday(&t2, NULL);
    // printf("Time of starting: %lf \n", TVAL(t2));
    // printf("Computing took %f ms\n", TDIFF(t1, t2));

    for (int i = 0; i < n; ++i) {
        switch(job[i].type){
            case dt_chr:
                hipHostFree(h_data[i].chr_data);
                hipHostFree(h_result[i].chr_data);
                break;
            case dt_int:
                hipHostFree(h_data[i].int_data);
                hipHostFree(h_result[i].int_data);
                break;
            case dt_flt:
                hipHostFree(h_data[i].flt_data);
                hipHostFree(h_result[i].flt_data);
            break;
        }
    }

    return 0;
}

