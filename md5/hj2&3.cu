#include "hip/hip_runtime.h"
// Memory object sizes:
// 1. hash table build: 2 * 8 * RLEN + 2 * 32 * 1024 * RBUCKETS
// 2. after hash_build before hash_join: 8 * RLEN
// 3. each hash_join: 8 * S_CHUNK_LEN + 8 * RLEN + 8 * n_results
#include <stdio.h>
#include <stdlib.h>

#include "hj.cu"
// #include "hj_kernels.cu"

#ifndef CUDA_SAFE_CALL
    #define CUDA_SAFE_CALL(call) \
        do { \
            hipError_t err = call; \
            if(hipSuccess != err) { \
                fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \
                        __FILE__, __LINE__, hipGetErrorString(err)); \
                exit(EXIT_FAILURE); \
            } \
        } while (0)
    #endif

//#define NR_BUCKETS_DEFAULT	256


// number of records in R
//#define RLEN		(40 * 1024L * 1024L)
#define RLEN		(10L * 1024L * 1024L)
// max of R's keys
#define RKEY_MAX	(1024 * 256)
// seed of R's keys
#define RKEY_SEED	1
// number of buckets for R's hash table; should not be larger than RKEY_MAX
#define RBUCKETS	(1024 * 8) // must be power of 2

// max of S's keys
#define SKEY_MAX	(1024 * 256)
// seed of S's keys
#define SKEY_SEED	2

// number of records in each chunk read from S
#define S_CHUNK_LEN		(64L * 1024L)
// how many chunks to be read from S
#define S_CHUNK_CNT		5

int read_r(record_t *r_tmp, int *rlen)
{
	// hipError_t res;

	// record_t *r_tmp = (record_t *)malloc(size_r);
	// if(!r_tmp) {
	// 	fprintf(stderr, "malloc failed for R\n");
	// 	return -1;
	// }
	// record_t *r_tmp;

	unsigned int seed = RKEY_SEED;
	for(int i = 0; i < RLEN; i++) {
		r_tmp[i].y = rand_r(&seed) % RKEY_MAX;
		r_tmp[i].x = i;
	}

	// *r = r_tmp;
	*rlen = RLEN;
	return 0;
}

// return the number of records actually read
int read_s(record_t *s, int slen, int skey_start)
{

	static unsigned int seed = SKEY_SEED;
	for(int i = 0; i < slen; i++) {
		s[i].y = rand_r(&seed) % (SKEY_MAX - skey_start) + skey_start;
		s[i].x = skey_start + i;
	}
	return slen;
}

// Assume R is the small table, upon which a hash table is built and kept in
// GPU memory. Assume S is the large table, for which data are fetched chunk
// by chunk, with one chunk, after another, joined with R.
// A problem with hash join is that, even though the joined results may be few,
// the number of buckets and sparse memory regions touched by the join may be
// plenty.
int main()
{
	hipStream_t *stream = (hipStream_t *) malloc(15 * sizeof(hipStream_t));

	record_t *h_r[2], *h_s[2][S_CHUNK_CNT];
	hash_table_t ht_r[2];
	int rlen, slen[2][S_CHUNK_CNT];
	struct timeval t1, t2, t_start, t_end;

	gettimeofday(&t_start, NULL);

	printf("Time of starting hj: %lf \n", TVAL(t_start));

	// Create cuda stream
        for (int i = 0; i < 15; ++i)
            CUDA_SAFE_CALL( hipStreamCreate(&stream[i]) );


	int build_hash_blocks = 64, build_hash_threads_per_block = 128;
	int scan_blocks = 512, scan_chunks;
	int scan_threads_per_block = 128;
	int scan_elems_per_block = 2 * scan_threads_per_block;
	int bytes_smem = sizeof(int) * scan_elems_per_block;


	// read r and build hash table
	gettimeofday(&t1, NULL);

	CUDA_SAFE_CALL(hipHostMalloc((void**)&h_r[0], sizeof(record_t) * RLEN));
	CUDA_SAFE_CALL(hipHostMalloc((void**)&h_r[1], sizeof(record_t) * RLEN));

	if(read_r(h_r[0], &rlen)) {
		fprintf(stderr, "failed to read r\n");
		return -1;
	}

	if(read_r(h_r[1], &rlen)) {
		fprintf(stderr, "failed to read r\n");
		return -1;
	}

	gettimeofday(&t2, NULL);
	printf("Time on reading R: %lf ms\n", TIME_DIFF(t1, t2));

	gettimeofday(&t1, NULL);

	
	// printf("Begin build_hash_table(r)\n");

	// varaibales for building hash table
	int *d_hist[2] = {NULL, NULL}, *d_loc[2] = {NULL, NULL};
	record_t *d_r[2] = {NULL, NULL};
	int ret = 0;


	for(int i = 0; i < 2; i++){
		ht_r[i].n_buckets = RBUCKETS;
		ht_r[i].d_rec = NULL;
		ht_r[i].d_idx = NULL;
		ht_r[i].n_records = rlen;
		if(!ht_r[i].n_buckets) {
			ht_r[i].n_buckets = NR_BUCKETS_DEFAULT;
		}
	}

	// for scan
	int *d_sumbuf[2];	// the buffer used to store sum updates across subarrays
	int *h_sumbuf[2];
	int sum_tot[2], sum_delta[2];



	// step 1: partition the array into many subarrays,
	// each of which is scanned separately
	scan_chunks = build_hash_blocks * build_hash_threads_per_block * ht_r[0].n_buckets / scan_elems_per_block;
	scan_chunks += (build_hash_blocks * build_hash_threads_per_block * ht_r[0].n_buckets % scan_elems_per_block) ? 1 : 0;
	scan_chunks = build_hash_blocks * build_hash_threads_per_block * ht_r[1].n_buckets / scan_elems_per_block;
	scan_chunks += (build_hash_blocks * build_hash_threads_per_block * ht_r[1].n_buckets % scan_elems_per_block) ? 1 : 0;

for(int i = 0; i < 2; i++){
	// copy records to GPU device memory
	CUDA_SAFE_CALL(hipMalloc((void **)&d_r[i], rlen * sizeof(record_t)));

	// build histogram matrix to collect how many
    // records each thread generates in each bucket
	CUDA_SAFE_CALL(hipMalloc((void **)&d_hist[i],
	        sizeof(int) * build_hash_blocks * build_hash_threads_per_block * ht_r[i].n_buckets));

	// prefix sum to get the offsets
	CUDA_SAFE_CALL(hipMalloc((void **)&d_loc[i],
	        sizeof(int) * build_hash_blocks * build_hash_threads_per_block * ht_r[i].n_buckets));

	// build the hash table
	CUDA_SAFE_CALL(hipMalloc((void **)&ht_r[i].d_rec, rlen * sizeof(record_t)));


	CUDA_SAFE_CALL(hipMalloc((void **)&ht_r[i].d_idx, (ht_r[i].n_buckets + 1) * sizeof(int)));


		CUDA_SAFE_CALL(hipMalloc((void **)&d_sumbuf[i], sizeof(int) * scan_chunks));

		// printf("scan: begin hipHostMalloc\n");
		CUDA_SAFE_CALL(hipHostMalloc((void**)&h_sumbuf[i], sizeof(int) * scan_chunks));
		// printf("scan: finish hipHostMalloc\n");
}


for(int i = 0; i < 2; i++){
    // printf("build_hash_table: begin hipMemcpyAsync(r)\n");
    CUDA_SAFE_CALL(hipMemcpyAsync(d_r[i], h_r[i], rlen * sizeof(record_t), hipMemcpyHostToDevice, stream[i]));
}
    // printf("build_hash_table: finish hipMemcpyAsync(r)\n");

for(int i = 0; i < 2; i++){
	hash_build_hist<<<build_hash_blocks, build_hash_threads_per_block, 0, stream[i]>>>(d_hist[i], d_r[i], rlen,
	        ht_r[i].n_buckets);
	// printf("build_hash_table: finish hash_build_hist\n");
	if(hipStreamSynchronize(stream[i]) != hipSuccess) {
		fprintf(stderr, "kernel failed at hash_build_hist\n");
		ret = -1;
		goto failed;
	}
	

	// printf("build_hash_table: begin scan\n");


	// printf("scan: begin prefix_sum\n");
	prefix_sum<<<scan_blocks, scan_threads_per_block, bytes_smem, stream[i]>>>(
	        d_loc[i], d_sumbuf[i], d_hist[i], scan_chunks, build_hash_blocks * build_hash_threads_per_block * ht_r[i].n_buckets);
	// printf("scan: finish prefix_sum\n");
	// printf("scan: begin hipDeviceSynchronize\n");
	if(hipStreamSynchronize(stream[i]) != hipSuccess) {
		fprintf(stderr, "kernel failed at prefix_sum\n");
		goto failed;
	}
}
	// printf("scan: finish hipDeviceSynchronize\n");

	// free(h_sumbuf);
	// hipFree(d_sumbuf);

	// step 2: update all scanned subarrays to derive the final result
	// res = hipMemcpy(h_sumbuf, d_sumbuf, sizeof(int) * nr_chunks,
	//         hipMemcpyDeviceToHost);
for(int i = 0; i < 2; i++){
	// printf("scan: begin hipMemcpyAsync\n");
	CUDA_SAFE_CALL(hipMemcpyAsync(h_sumbuf[i], d_sumbuf[i], sizeof(int) * scan_chunks,
	        hipMemcpyDeviceToHost, stream[i]));
	// printf("scan: finish hipMemcpyAsync\n");
}

for(int j = 0; j < 2; j++){	
	sum_tot[j] = 0;
	sum_delta[j] = h_sumbuf[j][0];
	for(int i = 1; i < scan_chunks; i++) {
		sum_tot[j] += sum_delta[j];
		sum_delta[j] = h_sumbuf[j][i];
		h_sumbuf[j][i] = sum_tot[j];
	}
	h_sumbuf[j][0] = 0;
	sum_tot[j] += sum_delta[j];
}

for(int i = 0; i < 2; i++){
	// printf("scan: begin hipMemcpyAsync\n");
	CUDA_SAFE_CALL(hipMemcpyAsync(d_sumbuf[i], h_sumbuf[i], sizeof(int) * scan_chunks,
	        hipMemcpyHostToDevice, stream[i]));
}
	// printf("scan: finish hipMemcpyAsync\n");

for(int i = 0; i < 2; i++){
	// printf("scan: begin prefix_sum_update\n");
	prefix_sum_update<<<scan_blocks, scan_threads_per_block, 0, stream[i]>>>(d_loc[i], d_sumbuf[i],
	        scan_chunks, build_hash_blocks * build_hash_threads_per_block * ht_r[i].n_buckets);
	// printf("scan: finish prefix_sum_update\n");
	// printf("scan: begin hipDeviceSynchronize\n");
	if(hipStreamSynchronize(stream[i]) != hipSuccess) {
		fprintf(stderr, "kernel failed at prefix_sum_update\n");
		goto failed;
	}
	// printf("scan: finish hipDeviceSynchronize\n");

	hash_build<<<build_hash_blocks, build_hash_threads_per_block, 0, stream[i]>>>(ht_r[i].d_rec, ht_r[i].d_idx,
	        d_r[i], rlen, d_loc[i], ht_r[i].n_buckets);
	if(hipStreamSynchronize(stream[i]) != hipSuccess) {
		fprintf(stderr, "kernel failed at hash_build\n");
		ret = -1;
		goto failed;
	}
}
	goto finish;

failed:
	free_hash_table(&ht_r[0]);
	free_hash_table(&ht_r[1]);
	// printf("scan: free\n");
	hipFree(h_sumbuf);
	// printf("scan: cudafree\n");
	hipFree(d_sumbuf);

finish:
	CUDA_FREE(d_r);
	CUDA_FREE(d_hist);
	CUDA_FREE(d_loc);
	// printf("scan: free\n");
	hipFree(h_sumbuf);
	// printf("scan: cudafree\n");
	hipFree(d_sumbuf);


// printf("build_hash_table: finish scan\n");
	CUDA_FREE(d_hist);


	hipFree(h_r);	// table R on the host is not needed any more

	gettimeofday(&t2, NULL);
	printf("Time on building hash table for R: %lf ms\n", TIME_DIFF(t1, t2));


	// for each chunk of s, join with r
	// h_s = (record_t *)malloc(sizeof(record_t) * S_CHUNK_LEN);
	// if(!h_s) {
	// 	fprintf(stderr, "malloc failed for s\n");
	// 	free_hash_table(&ht_r);
	// 	return -1;
	// }

	gettimeofday(&t1, NULL);
for(int k = 0; k < 2; k++){
	for(int i = 0; i < S_CHUNK_CNT; i++){
		CUDA_SAFE_CALL(hipHostMalloc((void**)&h_s[k][i], sizeof(record_t) * S_CHUNK_LEN));
		slen[k][i] = read_s(h_s[k][i], S_CHUNK_LEN, 0);
	}
}
	gettimeofday(&t2, NULL);
	printf("Time on reading S: %lf ms ( %lf ms per join )\n", TIME_DIFF(t1, t2), TIME_DIFF(t1, t2)/S_CHUNK_CNT);


	record_t *h_z[S_CHUNK_CNT];
	int zlen[S_CHUNK_CNT];

	gettimeofday(&t1, NULL);
	for(int i = 0; i < S_CHUNK_CNT; i++){
		CUDA_SAFE_CALL(hipHostMalloc((void**)&h_z[i], sizeof(record_t) * S_CHUNK_LEN));
		zlen[i] = read_s(h_z[i], S_CHUNK_LEN, 0);
	}
	gettimeofday(&t2, NULL);
	printf("Time on reading Z: %lf ms ( %lf ms per join )\n", TIME_DIFF(t1, t2), TIME_DIFF(t1, t2)/S_CHUNK_CNT);


	// The number of result records joined per chunk is approximately:
	// RLEN * S_CHUNK_LEN / max(RKEY_MAX, SKEY_MAX)
	gettimeofday(&t1, NULL);

	for(int i = 0; i < S_CHUNK_CNT; i++) {
		// printf("%d\n", i);
		// join with r
		if(slen[0][i] > 0) {
			// printf("Begin hash_join\n");
			if(hash_join(NULL, NULL, &ht_r[0], h_s[0][i], slen[0][i], stream, i)) {
				fprintf(stderr, "hash join failed for the %dth chunk of S\n",
				        i);
				break;
			}
			// printf("Finish hash_join\n");
		}

		if(slen[1][i] > 0) {
			// printf("Begin hash_join\n");
			if(hash_join(NULL, NULL, &ht_r[1], h_s[1][i], slen[1][i], stream, S_CHUNK_CNT+i)) {
				fprintf(stderr, "hash join failed for the %dth chunk of S\n",
				        i);
				break;
			}
			for(int j = 0; j < S_CHUNK_CNT; j++) {
				if(zlen[j] > 0) {
					if(hash_join(NULL, NULL, &ht_r[1], h_z[j], zlen[j], stream, S_CHUNK_CNT+i)) {
						fprintf(stderr, "hash join failed for the %dth chunk of Z\n", j);
						break;
					}
				}
			}
		}
	}
	gettimeofday(&t2, NULL);
	printf("Time on hash join: %lf ms ( %lf ms per join )\n", TIME_DIFF(t1, t2), TIME_DIFF(t1, t2)/S_CHUNK_CNT);

	free_hash_table(&ht_r[0]);
	free_hash_table(&ht_r[1]);
	hipFree(h_s);

	gettimeofday(&t_end, NULL);
	printf("Total time taken: %lf ms\n", TIME_DIFF(t_start, t_end));
	printf("Time of ending hj: %lf \n", TVAL(t_end));
	return 0;
}
